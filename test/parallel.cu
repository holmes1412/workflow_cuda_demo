#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 9

__global__ void kernel(float *a, float *b, float *c, int batch_size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < batch_size)
		c[i] = a[i] + b[i];
}

int main()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	float a[N] = {0.1, 0.2, 0.3, 0.4, 0.1, 0.2, 0.3, 0.4, 0.5};
	float b[N] = {0.1, 0.2, 0.3, 0.4, 0.1, 0.2, 0.3, 0.4, 0.5};
	float c[N];

	hipStream_t *streams = new hipStream_t[deviceCount];
/*
	float **d_a = new float*[deviceCount];
	float **d_b = new float*[deviceCount];
	float **d_c = new float*[deviceCount];
*/

	// make device data ptr for each GPU
	float *d_a[deviceCount];
	float *d_b[deviceCount];
	float *d_c[deviceCount];

	int batch_size = (N + deviceCount - 1)/ deviceCount;

	// create stream for each GPU and malloc the device memory
	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipStreamCreate(&streams[i]);
		hipMalloc(&d_a[i], batch_size * sizeof(float));
		hipMalloc(&d_b[i], batch_size * sizeof(float));
		hipMalloc(&d_c[i], batch_size * sizeof(float));
	}

	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipMemcpyAsync(d_a[i], &a[i * batch_size], batch_size * sizeof(float),
						hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(d_b[i], &b[i * batch_size], batch_size * sizeof(float),
						hipMemcpyHostToDevice, streams[i]);
	}

	dim3 grid = 1;
	dim3 block = batch_size;
	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		kernel<<<grid, block, 0, streams[i]>>>(d_a[i], d_b[i], d_c[i],
											   batch_size);
	}

	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipMemcpyAsync(&c[i * batch_size], d_c[i], batch_size * sizeof(float),
						hipMemcpyDeviceToHost, streams[i]);
	}

	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipStreamSynchronize(streams[i]);
		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
			printf("stream-%d error: %s\n", i, hipGetErrorString(error));
	}
	
	for (int i = 0; i < N; ++i)
		printf("c[%d]=%f\n", i, c[i]);

	for (int i = 0; i < deviceCount; ++i)
	{
		hipSetDevice(i);
		hipFree(d_a[i]);
		hipFree(d_b[i]);
		hipFree(d_c[i]);
		hipStreamDestroy(streams[i]);
	}

	delete[] streams;

	return 0;
}

